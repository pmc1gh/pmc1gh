/**
 * CUDA Point Alignment
 * George Stathopoulos, Jenny Lee, Mary Giambrone, 2019*/

#include <cstdio>
#include <stdio.h>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "hip/hip_runtime_api.h"
#include <string>
#include <fstream>

#include "obj_structures.h"

// hip/hip_runtime_api.h contains the error checking macros. note that they're called
// CUDA_CALL, CUBLAS_CALL, and CUSOLVER_CALL instead of the previous names

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char *argv[]) {

    if (argc != 4)
    {
        printf("Usage: ./point_alignment [file1.obj] [file2.obj] [output.obj]\n");
        return 1;
    }

    std::string filename, filename2, output_filename;
    filename = argv[1];
    filename2 = argv[2];
    output_filename = argv[3];

    std::cout << "Aligning " << filename << " with " << filename2 <<  std::endl;
    Object obj1 = read_obj_file(filename);
    std::cout << "Reading " << filename << ", which has " << obj1.vertices.size() << " vertices" << std::endl;
    Object obj2 = read_obj_file(filename2);

    std::cout << "Reading " << filename2 << ", which has " << obj2.vertices.size() << " vertices" << std::endl;
    if (obj1.vertices.size() != obj2.vertices.size())
    {
        printf("Error: number of vertices in the obj files do not match.\n");
        return 1;
    }

    ///////////////////////////////////////////////////////////////////////////
    // Loading in obj into vertex Array
    ///////////////////////////////////////////////////////////////////////////

    int point_dim = 4; // 3 spatial + 1 homogeneous
    int num_points = obj1.vertices.size();

    // in col-major
    float * x1mat = vertex_array_from_obj(obj1);
    float * x2mat = vertex_array_from_obj(obj2);

    ///////////////////////////////////////////////////////////////////////////
    // Point Alignment
    ///////////////////////////////////////////////////////////////////////////

    // TODO: Initialize cublas handle
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    float * dev_x1mat;
    float * dev_x2mat;
    float * dev_xx4x4;
    float * dev_x1Tx2;

    // TODO: Allocate device memory and copy over the data onto the device
    // Hint: Use hipblasSetMatrix() for copying

    hipMalloc((void **) &dev_x1mat, num_points * point_dim * sizeof(float));
    hipMalloc((void **) &dev_x2mat, num_points * point_dim * sizeof(float));
    hipMalloc((void **) &dev_xx4x4, point_dim * point_dim * sizeof(float));
    hipMalloc((void **) &dev_x1Tx2, point_dim * point_dim * sizeof(float));

    hipblasSetMatrix(num_points, point_dim, sizeof(float), x1mat, num_points,
                    dev_x1mat, num_points);
    hipblasSetMatrix(num_points, point_dim, sizeof(float), x2mat, num_points,
                    dev_x2mat, num_points);

    // Now, proceed with the computations necessary to solve for the linear
    // transformation.

    float one = 1;
    float zero = 0;

    // TODO: First calculate xx4x4 and x1Tx2
    // Following two calls should correspond to:
    //   xx4x4 = Transpose[x1mat] . x1mat
    //   x1Tx2 = Transpose[x1mat] . x2mat

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, point_dim, point_dim,
                num_points, &one, dev_x1mat, num_points, dev_x1mat, num_points,
                &zero, dev_xx4x4, point_dim);
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, point_dim, point_dim,
                num_points, &one, dev_x1mat, num_points, dev_x2mat, num_points,
                &zero, dev_x1Tx2, point_dim);

    // TODO: Finally, solve the system using LU-factorization! We're solving
    //         xx4x4 . m4x4mat.T = x1Tx2   i.e.   m4x4mat.T = Inverse[xx4x4] . x1Tx2
    //
    //       Factorize xx4x4 into an L and U matrix, ie.  xx4x4 = LU
    //
    //       Then, solve the following two systems at once using cusolver's getrs
    //           L . temp  =  P . x1Tx2
    //       And then then,
    //           U . m4x4mat = temp
    //
    //       Generally, pre-factoring a matrix is a very good strategy when
    //       it is needed for repeated solves.

    // TODO: Make handle for cuSolver
    hipsolverHandle_t solver_handle;

    hipsolverDnCreate(&solver_handle);

    // TODO: Initialize work buffer using hipsolverDnSgetrf_bufferSize
    float * work;
    int Lwork;

    hipsolverDnSgetrf_bufferSize(solver_handle, point_dim, point_dim, dev_xx4x4,
                                point_dim, &Lwork);

    // TODO: compute buffer size and prepare memory

    hipMalloc((void **) &work, Lwork * sizeof(float));

    // TODO: Initialize memory for pivot array, with a size of point_dim
    int * pivots;

    hipMalloc((void **) &pivots, point_dim * sizeof(int));


    int *info;

    hipMalloc((void **) &info, sizeof(int));


    // TODO: Now, call the factorizer hipsolverDnSgetrf, using the above initialized data

    hipsolverDnSgetrf(solver_handle, point_dim, point_dim, dev_xx4x4, point_dim,
                     work, pivots, info);

    // TODO: Finally, solve the factorized version using a direct call to hipsolverDnSgetrs

    hipsolverDnSgetrs(solver_handle, HIPBLAS_OP_N, point_dim, point_dim,
                     dev_xx4x4, point_dim, pivots, dev_x1Tx2, point_dim, info);


    // TODO: Destroy the cuSolver handle

    hipsolverDnDestroy(solver_handle);


    // TODO: Copy final transformation back to host. Note that at this point
    // the transformation matrix is transposed
    float * out_transformation;

    out_transformation =
        (float *) malloc(point_dim * point_dim * sizeof(float));

    hipMemcpy(out_transformation,
               dev_x1Tx2, point_dim * point_dim * sizeof(float),
               hipMemcpyDeviceToHost);

    // TODO: Don't forget to set the bottom row of the final transformation
    //       to [0,0,0,1] (right-most columns of the transposed matrix)

    out_transformation[3*point_dim] = 0;
    out_transformation[3*point_dim+1] = 0;
    out_transformation[3*point_dim+2] = 0;
    out_transformation[3*point_dim+3] = 1;

    // Print transformation in row order.
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            std::cout << out_transformation[i * point_dim + j] << " ";
        }
        std::cout << "\n";
    }

    ///////////////////////////////////////////////////////////////////////////
    // Transform point and print output object file
    ///////////////////////////////////////////////////////////////////////////

    // TODO Allocate and Initialize data matrix
    float * dev_pt;

    hipMalloc((void **) &dev_pt, num_points * point_dim * sizeof(float));
    hipblasSetMatrix(num_points, point_dim, sizeof(float), x1mat, num_points,
                    dev_pt, num_points);

    // TODO Allocate and Initialize transformation matrix
    float * dev_trans_mat;

    hipMalloc((void **) &dev_trans_mat, point_dim * point_dim * sizeof(float));
    hipblasSetMatrix(point_dim, point_dim, sizeof(float), out_transformation,
                    point_dim, dev_trans_mat, point_dim);

    // TODO Allocate and Initialize transformed points
    float * dev_trans_pt;

    hipMalloc((void **) &dev_trans_pt, point_dim * num_points * sizeof(float));

    float one_d = 1;
    float zero_d = 0;

    // TODO Transform point matrix
    //          (4x4 trans_mat) . (nx4 pointzx matrix)^T = (4xn transformed points)

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, point_dim, num_points,
                point_dim, &one_d, dev_trans_mat, point_dim, dev_pt, num_points,
                &zero_d, dev_trans_pt, point_dim);

    // So now dev_trans_pt has shape (4 x n)
    float * trans_pt;

    trans_pt = (float *) malloc(point_dim * num_points * sizeof(float));
    hipblasGetMatrix(point_dim, num_points, sizeof(float), dev_trans_pt,
                    point_dim, trans_pt, point_dim);

    // get Object from transformed vertex matrix
    Object trans_obj = obj_from_vertex_array(trans_pt, num_points, point_dim,
                                             obj1);

    // print Object to output file
    std::ofstream obj_file (output_filename);
    print_obj_data(trans_obj, obj_file);

    // free CPU memory
    free(trans_pt);

    ///////////////////////////////////////////////////////////////////////////
    // Free Memory
    ///////////////////////////////////////////////////////////////////////////

    // TODO: Free GPU memory

    hipFree(dev_x1mat);
    hipFree(dev_x2mat);
    hipFree(dev_xx4x4);
    hipFree(dev_x1Tx2);

    hipFree(work);
    hipFree(pivots);
    hipFree(info);

    hipFree(dev_pt);
    hipFree(dev_trans_mat);
    hipFree(dev_trans_pt);

    hipblasDestroy(handle);

    // TODO: Free CPU memory
    free(out_transformation);
    free(x1mat);
    free(x2mat);

}
